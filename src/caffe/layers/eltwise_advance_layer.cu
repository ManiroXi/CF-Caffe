#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Copyright (c) 2017
// The Chinese University of Hong Kong
// Written by Hu Xiaowei
//
// EltwiseAdvLayer the bottom[1] has lower dims than bottom[0].
// Only the Production operation was implemented.
// ------------------------------------------------------------------

#include <cfloat>
#include <vector>

#include "caffe/layers/eltwise_advance_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype> 
__global__ void MultiForward(const int nthreads, const int replicate_times, const Dtype* bottom_data, const Dtype* bottom_producter, Dtype* top_data, const int channel, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {

     int w = index % width;
     int h = (index / width) % height;
     int c = (index / width / height) % channel;
     int n = index / width / height / channel;

     top_data[index] = bottom_data[index] * bottom_producter[((n*channel + (c/replicate_times))*height + h)*width + w];
    
  }
}


template <typename Dtype> 
__global__ void MultiBackward(const int nthreads, const int replicate_times, const Dtype* top_diff, const Dtype* bottom_data0, const Dtype* bottom_data1, Dtype* bottom_diff0, Dtype* bottom_diff1, const int channel, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {

     int w = index % width;
     int h = (index / width) % height;
     int c = (index / width / height) % channel;
     int n = index / width / height / channel;

     bottom_diff1[index] = 0;

     for (int i=0; i<replicate_times; i++)
     {
         bottom_diff0[((n*channel + (replicate_times*c+i))*height + h)*width + w] = bottom_data1[index] * top_diff[((n*channel + (replicate_times*c+i))*height + h)*width + w]; //another_bottom_data*top_diff

        bottom_diff1[index] += bottom_data0[((n*channel + (replicate_times*c+i))*height + h)*width + w] * top_diff[((n*channel + (replicate_times*c+i))*height + h)*width + w];
    
     }
    
  }
}

template <typename Dtype>
void EltwiseAdvLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_producter = bottom[1]->gpu_data();
  const int replicate_times = int(bottom[0]->count()) / int(bottom[1]->count());

  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    MultiForward<Dtype>  
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, replicate_times, bottom_data, bottom_producter, top_data, channels_, height_, width_);
  CUDA_POST_KERNEL_CHECK;
    break;
  case EltwiseParameter_EltwiseOp_SUM:
    NOT_IMPLEMENTED;
    break;
  case EltwiseParameter_EltwiseOp_MAX:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}

template <typename Dtype>
void EltwiseAdvLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
 
  const int count = bottom[1]->count();
  const Dtype* top_diff = top[0]->gpu_diff();

  Dtype* bottom_diff0 = bottom[0]->mutable_gpu_diff();
  Dtype* bottom_diff1 = bottom[1]->mutable_gpu_diff();

  const Dtype* bottom_data0 = bottom[0]->gpu_data();
  const Dtype* bottom_data1 = bottom[1]->gpu_data(); 

  const int replicate_times = int(bottom[0]->count()) / int(bottom[1]->count());

  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    if (stable_prod_grad_) {

       MultiBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
           <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, replicate_times, top_diff, bottom_data0, bottom_data1, bottom_diff0, bottom_diff1, channels_, height_, width_);
      CUDA_POST_KERNEL_CHECK;
    
    } else {
      NOT_IMPLEMENTED;
    }
    break;
  case EltwiseParameter_EltwiseOp_SUM:
    NOT_IMPLEMENTED;
    break;
  case EltwiseParameter_EltwiseOp_MAX:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EltwiseAdvLayer);

}  // namespace caffe
