#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_loss_layer_BER_weight.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void SigmoidCrossEntropyLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* target, Dtype* loss,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts, float weight0_, float weight1_, float weight_a0_, float weight_a1_, int num_pixel_) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const float target_value = static_cast<float>(target[i]);

    if (has_ignore_label_ && static_cast<int>(target_value) == ignore_label_) {
      loss[i] = 0;
      counts[i] = 0;
    } else {
      loss[i] = input_data[i] * (target[i] - (input_data[i] >= 0)) -
          log(1 + exp(input_data[i] - 2 * input_data[i] *
          (input_data[i] >= 0)));
      counts[i] = 1;

      // weighted
      if (target_value>0.5) //positive
      { loss[i] *= (weight_a0_ + weight0_); }
      else
      { loss[i] *= (weight_a1_ + weight1_); }

      //printf("(weight_a0_ + weight0_): %f, (weight_a1_ + weight1_): %f, After_loss[i]:%f, target_value:%f\n",(weight_a0_ + weight0_), (weight_a1_ + weight1_), loss[i], target_value);
    }
  }
}

template <typename Dtype>
__global__ void SigmoidCrossEntropyLossIgnoreDiffGPU(const int count,
    const int ignore_label, const Dtype* target, Dtype* diff) {
  CUDA_KERNEL_LOOP(i, count) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value == ignore_label) {
      diff[i] = 0;
    }
  }
}


template <typename Dtype>
void SigmoidCrossEntropyLossBERweightLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* count_data = bottom[1]->mutable_gpu_diff();
  Dtype valid_count;


  // Hu Xiaowei 2017
  // BER = 1-1/2*(TP/(TP+FN)+TN/(TN+FP)) 
  // Calculate the BER error and cross_entropy_loss with weight (area) for each image and use it as a weight
  // 1/2 + 1/2
  //int h = bottom[0]->shape(1);
  //int w = bottom[0]->shape(2);

  const Dtype* input_data_cpu = bottom[0]->cpu_data();
  const Dtype* target_cpu = bottom[1]->cpu_data();
  
  const int num_image = bottom[0]->shape(0);
  int num_pixel = bottom[0]->count(1);

  float weight0, weight1; // [batch_size=1] [weight_p, weight_n];
  float weight_a0, weight_a1;

  for (int i=0; i<num_image; i++)
  {
      int countPos = 0;
      int countNeg = 0;
      int countTP = 0; //true positive;
      int countTN = 0; //true negative;

      for (int j=0; j<num_pixel; j++)
      {   
          float t_data = target_cpu[i*num_pixel+j];
          float i_data = input_data_cpu[i*num_pixel+j];


         if (t_data>0.5) //positive
          {  
             countPos++;
             if (i_data>0.5) //positive
             {
                countTP++;
             }
          }
          else
          {  
             countNeg++;
             if (i_data<=0.5) //negative
             {
                countTN++;
             }
          }
      } 

      weight_a0 = (float)countNeg / (float)(countPos+countNeg);
      weight_a1 = (float)countPos / (float)(countPos+countNeg);

      if (countPos==0)
      { weight0 = 1; }
      else
      { weight0 = 1-((float)countTP/(float)countPos); }//positive weight
      if (countNeg==0)
      { weight1 = 1; }
      else
      { weight1 = 1-((float)countTN/(float)countNeg); }//negative weight
  }  

  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidCrossEntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, input_data, target, loss_data,
      has_ignore_label_, ignore_label_, count_data, weight0, weight1, weight_a0, weight_a1, num_pixel);
  // Only launch another CUDA kernel if we actually need the valid count.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(count, count_data, &valid_count);
  } else {
    valid_count = count;
  }
  Dtype loss;
  caffe_gpu_asum(count, loss_data, &loss);
  normalizer_ = get_normalizer(normalization_, valid_count);
  top[0]->mutable_cpu_data()[0] = loss / normalizer_;
}

template <typename Dtype>
void SigmoidCrossEntropyLossBERweightLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
    // Zero out gradient of ignored targets.
    if (has_ignore_label_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidCrossEntropyLossIgnoreDiffGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, ignore_label_, target, bottom_diff);
    }
    // Scale down gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer_;
    caffe_gpu_scal(count, loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossBERweightLayer);

}  // namespace caffe
